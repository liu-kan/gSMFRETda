#include "hip/hip_runtime.h"
#include "cuda_tools.hpp"
#include "mc.hpp"
#include <cstddef>
#include <string>
#include <thread>
#include <time.h>
#include <chrono>
#define VECTOR_SIZE 64
#include "binom.cuh"
#include "gen_rand.cuh"
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <numeric>
#include "tools.hpp"
#include <boost/uuid/uuid.hpp>
#include <boost/uuid/uuid_generators.hpp>
#include <hip/hip_runtime_api.h>

void CUDART_CB myStreamCallback(hipStream_t stream, hipError_t status,
                                void *data) {
    if (status) {
        std::cout << "Error: " << hipGetErrorString(status) << "-->";
    }
}

int showGPUsInfo(int dn, char *gpuuid, int *streamCount) {
    int nDevices, i, n_Devices;
    CUDA_CHECK_RETURN(hipGetDeviceCount(&nDevices));
    if (dn >= 0) {
        n_Devices = dn + 1;
        i = dn;
    } else {
        i = 0;
        n_Devices = nDevices;
    }
    if (dn < nDevices) {
        int pgid;
        hipGetDevice( &pgid );
        for (; i < n_Devices; i++) {
            hipDeviceProp_t prop;
            CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, i));
            printf("Device Index: %d\n", i);
            if (gpuuid) {
#if (CUDART_VERSION < 10000)
                s::uuid a_uuid = s::random_generator()();
                memcpy(gpuuid, &a_uuid, 16);
#else
                memcpy(gpuuid, &(prop.uuid.bytes[0]), 16);
#endif
            }
#if (CUDART_VERSION >= 10000)
            printf("  Device UUID: ");
            for (int i = 0; i < 16; i++) {
                printf("%hhx", prop.uuid.bytes[i]);
            }
#endif
            printf("\n  Concurrent copy and kernel execution: \t %s with "
                   "%d copy "
                   "engine(s)\n",
                   (prop.deviceOverlap ? "Yes" : "No"), prop.asyncEngineCount);
            if (streamCount) {
                if (!prop.deviceOverlap)
                    *streamCount = 1;
                else
                    *streamCount = prop.asyncEngineCount * 2;
            }
            printf("  Device name: %s\n", prop.name);
            // printf("  GPU global memory = %lu GBytes\n",
            //        (prop.totalGlobalMem >> 30) + 1);
            hipSetDevice( i );
            size_t free, total;
            hipMemGetInfo( &free, &total );
            cout << "GPU " << i << " memory: free=" << addThousandSeparators(std::to_string((free>>20))) << " MB, total=" 
                << addThousandSeparators(std::to_string((total>>20))) <<" MB"<< std::endl;
        }
        hipSetDevice( pgid );
    }
    return nDevices;
}
#define CUDAstream_CHECK_LAST_ERROR                                                 \
    hipStreamAddCallback(streams[sid], myStreamCallback, nullptr, 0)

__global__ void
mc_kernel(int64_t *start, int64_t *stop, int64_t *g_burst_ad, int64_t *g_burst_dd,
          int64_t *g_istart, int *g_phCount, float *T, /*float* SgDivSr,*/
          float clk_p, float bg_ad_rate, float bg_dd_rate, float *gpe, float *gpv,
          float *gpk, float* gpp_i2j, float *gpp, float *P_i2j, int N, int s_n,
          hiprandStateScrambledSobol64 *devQStates, rk_state *devStates, retype *mcE,
          int reSampleTimes, unsigned char debug = 0, float gamma = 0.34,
          float beta = 1.42, float DexDirAem = 0.08, float Dch2Ach = 0.07,
          float r0 = 52) {
    int NN = N * reSampleTimes;
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tidx < NN) {
        // idx is burst id
        // int idx=tidx;//%reSampleTimes;
        // int idx = __fdiv_rd(tidx,reSampleTimes);
        // int sampleTime=tidx%reSampleTimes;
        // If n is a power of 2, ( i / n ) is equivalent to ( i ≫ log2 n ) and (
        // i % n ) is equivalent to ( i & n - 1 ).
        int idx = tidx >> ((int)log2((double)reSampleTimes));
        // int sampleTime = tidx & reSampleTimes - 1;
        int phCount = g_phCount[idx];
        int64_t *burst_ad = g_burst_ad + g_istart[idx];
        int64_t *burst_dd = g_burst_dd + g_istart[idx];
        float F = 0;
        mcE[tidx] = 0;
        // for (int sampleTime=0;sampleTime<reSampleTimes;sampleTime++){
        // // int sampleTime=tidx/reSampleTimes;
        int si = drawDisIdx(s_n, gpp, devQStates + tidx);
        float mcSpendTime = 0;
        matXfMapper matKmp(gpk, s_n, s_n);
        int count = 0;
        int64_t bin0clk_t = start[idx];
        int64_t bin1clk_t = start[idx];
        int sj = 0, binIdxStart = 0;
        while (T[idx] > mcSpendTime) {
            sj = drawJ_Si2Sj(P_i2j + tidx * s_n, gpp_i2j, s_n, si, devQStates + tidx);
            // if(si==0 && sj==2&& tidx<100)
            //     printf("s_n=%d,sj=%d,gpp=%f, %f, %f,
            //     tidx=%d\n",s_n,sj,gpp[0],gpp[1],gpp[2],tidx);
            float st = drawTau(matKmp(sj, si), devQStates + tidx, 0);
            // if(si==0 && sj==2&& tidx<100)
            //     printf("drawTau=%f\n",st);
            mcSpendTime = mcSpendTime + st;
            // si=sj;
            if (mcSpendTime >= T[idx]) {
                //     bins.append(stop[idx]);
                bin1clk_t = stop[idx];
            } else {
                //     bins.append(*(bins.at(0))+mcSpendTime/clk_p);
                bin1clk_t = bin0clk_t + mcSpendTime / clk_p;
            }
            // [bin0clk_t bin1clk_t) is the clk timing range,
            // Then try to get the ad and dd count in this range.
            bool sdd = false, sad = false;//, bdd = false, bad = false;
            int f_id = 0, f_ia = 0;
            int64_t ddx, adx;
            long ai = 0;
            for (int iinb = binIdxStart; iinb < phCount; iinb++) {
                ddx = burst_dd[iinb];
                adx = burst_ad[iinb];
                // if(debug)
                // if(idx==200)
                //     printf("ddx= %ld, adx= %ld\n",ddx,adx);

                if (ddx >= bin1clk_t || adx >= bin1clk_t || iinb == phCount - 1) {
                    binIdxStart = iinb;
                    // calac F
                    float f_if = (gamma - Dch2Ach) * f_id + (1 - DexDirAem) * f_ia;
                    float f_i = 0;
                    F += f_if;
                    if (bg_dd_rate < 1e-4) {
                        f_i = floorf(f_if + 0.5);
                    } else {
                        float t_diff = (bin1clk_t - bin0clk_t) * clk_p;
                        float rf_ia = (1 - DexDirAem) * f_ia;
                        float bg_a;
                        draw_P_B_Tr(&bg_a, &rf_ia, 1, &t_diff, bg_ad_rate,
                                    devQStates + tidx);
                        float rf_id = (gamma - Dch2Ach) * f_id;
                        float bg_d;
                        draw_P_B_Tr(&bg_d, &rf_id, 1, &t_diff, bg_dd_rate,
                                    devQStates + tidx);
                        f_i = floorf(f_if - bg_d - bg_a + 0.5);
                    }
                    float de = drawE(gpe[si], r0, gpv[si], devQStates + tidx);
                    ai = rk_binomial(devStates + tidx, f_i, de);
                    break;
                }
                if (ddx > 0)
                    sdd = true;
                if (adx > 0)
                    sad = true;
                if (sad && sdd)
                    continue;
                if (sad /*&& !bad*/)
                    f_ia++;
                if (sdd /*&& !bdd*/)
                    f_id++;
            }
            mcE[tidx] += ai;
            count++;
            bin0clk_t = bin1clk_t;
            si = sj;
#define __count__ 3
            if (debug)
                if (count > __count__)
                    printf("burst id %d trans %d > %d. clk_p= %g, mcE[%d]= %g\n",
                           idx, count, __count__, clk_p, tidx, mcE[tidx]);
        }
        if (F > 0)
            mcE[tidx] = mcE[tidx] / F;
        else
            mcE[tidx] = 0;
    }
}

/**
 * @brief  Actual Setup of GPUID, if you have multi gpu. 
 * According to [nvidia blog](https://developer.nvidia.com/blog/cuda-pro-tip-always-set-current-device-avoid-multithreading-bugs/) this fuction must *Always* be called when a new host threads need to call cu kernel.
 * 
 */
void mc::set_gpuid() {
    cout<<"Thread id: "<<std::this_thread::get_id()<<" hipSetDevice: "<<devid<<endl;
    CUDA_CHECK_RETURN(hipSetDevice(devid));
    if (profiler) {
        std::cout << "hipProfilerStart" << std::endl;
        CUDA_CHECK_RETURN(hipProfilerStart());
    }
}
mc::mc(int id, int _streamNum, unsigned char de, std::uintmax_t hdf5size,
       bool _profiler) {
    debug = de;
    data_gpu_inited=false;
    profiler = _profiler;
    devid = id;
    hipGetDeviceCount(&nDevices);
    if (_streamNum == 0) {
        nDevices = showGPUsInfo(devid, gpuuuid, &streamNum);
        printf("streamCount is automatically determined as %d\n", streamNum);
    } else {
        nDevices = showGPUsInfo(devid, gpuuuid);
        streamNum = _streamNum;
    }
    workerNum.store(streamNum);
    if (devid >= nDevices || devid < 0) {
        std::cout << "gpu id set error!" << std::endl;
        return;
    }
    set_gpuid();
    conversion_buff_sz=new int[streamNum];
    mr = new mrImp(hdf5size, 0.85, devid,false,0);
    // setAllocator("rmmDefaultPool");
    streams = (hipStream_t *)malloc(sizeof(hipStream_t) * streamNum);
    for (int sid = 0; sid < streamNum; sid++) {
        CUDA_CHECK_RETURN(
            hipStreamCreateWithFlags(&(streams[sid]), hipStreamNonBlocking));
        // CUDA_CHECK_RETURN(hipStreamCreateWithFlags (
        // &(streams[sid]),hipStreamDefault) );
        streamFIFO.push(sid);
    }
    matK = NULL;
    matP = NULL;
    // hpv=(float **)malloc(sizeof(float*)*streamNum);
    // hpk=(float **)malloc(sizeof(float*)*streamNum);
    // hpp=(float **)malloc(sizeof(float*)*streamNum);
    // hpe=(float **)malloc(sizeof(float*)*streamNum);
    g_conversion_buff = (int **)malloc(sizeof(int *) * streamNum);
    gpv = (float **)malloc(sizeof(float *) * streamNum);
    gpk = (float **)malloc(sizeof(float *) * streamNum);
    gpp_i2j = (float**)malloc(sizeof(float*) * streamNum);
    gpp = (float **)malloc(sizeof(float *) * streamNum);
    gpe = (float **)malloc(sizeof(float *) * streamNum);
    g_P_i2j = (float **)malloc(sizeof(float *) * streamNum);
    matK=(float**)malloc(sizeof(float *) * streamNum);
    matP_i2j = (float**)malloc(sizeof(float*) * streamNum);
    matP=(float**)malloc(sizeof(float *) * streamNum);
    oldN = new int[streamNum];
    std::fill_n(oldN, streamNum, 0);
    // std::memset(hpv, 0, sizeof(float*)*streamNum);
    // std::memset(hpk, 0, sizeof(float*)*streamNum);
    // std::memset(hpp, 0, sizeof(float*)*streamNum);
    // std::memset(hpe, 0, sizeof(float*)*streamNum);
    std::memset(matP, 0, sizeof(float *) * streamNum);
    std::memset(matK, 0, sizeof(float *) * streamNum);
    std::memset(matP_i2j, 0, sizeof(float*) * streamNum);
    std::memset(g_P_i2j, 0, sizeof(float *) * streamNum);
    std::memset(gpv, 0, sizeof(float *) * streamNum);
    std::memset(gpp_i2j, 0, sizeof(float *) * streamNum);
    std::memset(gpk, 0, sizeof(float*) * streamNum);
    std::memset(g_conversion_buff, 0, sizeof(int*) * streamNum);
    std::memset(gpp, 0, sizeof(float *) * streamNum);
    std::memset(gpe, 0, sizeof(float *) * streamNum);
    s_n = new int[streamNum];
    gridSize = new int[streamNum];
    begin_burst = new int[streamNum];
    end_burst = new int[streamNum];
    std::fill_n(s_n, streamNum, 0);
    std::fill_n(gridSize, streamNum, 0);
    std::fill_n(begin_burst, streamNum, 0);
    std::fill_n(end_burst, streamNum, 0);
    mcE = (retype **)malloc(sizeof(retype *) * streamNum);
    hmcE = (retype **)malloc(sizeof(retype *) * streamNum);
    std::memset(mcE, 0, sizeof(retype *) * streamNum);
    std::memset(hmcE, 0, sizeof(retype *) * streamNum);
    // hmcE=mcE=NULL;
    devStates = (rk_state **)malloc(sizeof(rk_state *) * streamNum);
    devQStates = (hiprandStateScrambledSobol64 **)malloc(
        sizeof(hiprandStateScrambledSobol64 *) * streamNum);
    // hostVectors64=(hiprandDirectionVectors64_t**)malloc(sizeof(hiprandDirectionVectors64_t*)*streamNum);
    // hostScrambleConstants64=(unsigned long long int**)malloc(sizeof(unsigned
    // long long int*)*streamNum);
    devDirectionVectors64 = (unsigned long long int **)malloc(
        sizeof(unsigned long long int *) * streamNum);
    devScrambleConstants64 = (unsigned long long int **)malloc(
        sizeof(unsigned long long int *) * streamNum);
    std::memset(devStates, 0, sizeof(rk_state *) * streamNum);
    std::memset(devQStates, 0, sizeof(hiprandStateScrambledSobol64 *) * streamNum);
    // std::memset(hostVectors64, 0,
    // sizeof(hiprandDirectionVectors64_t*)*streamNum);
    std::memset(devDirectionVectors64, 0,
                sizeof(unsigned long long int *) * streamNum);
    // std::memset(hostScrambleConstants64, 0, sizeof(unsigned long long
    // int*)*streamNum);
    std::memset(devScrambleConstants64, 0,
                sizeof(unsigned long long int *) * streamNum);
    reSampleTimes = 4;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, mc_kernel, 0, 0);
    // blockSize=128;
    printf("blockSize = %d\n", blockSize);
    CURAND_CALL(hiprandGetDirectionVectors64(
        &hostVectors64, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6));
    CURAND_CALL(hiprandGetScrambleConstants64(&hostScrambleConstants64));
}
void mc::set_reSampleTimes(int t) { reSampleTimes = pow(2, (int)log2(t)); }
hipStream_t* mc::getStreams(int *numStream =NULL) {
    if(numStream)
        *numStream=streamNum;
    return streams;
    
}

void mc::givebackStream(int i) { streamFIFO.push(i); }

bool mc::get_max_conversion_capacity(int max_stateNum){
    if(data_gpu_inited){
        double N=((double)sz_burst)*streamNum;
        size_t reserved_gpumem= sz_burst * reSampleTimes * (sizeof(rk_state)+ 
            sizeof(hiprandStateScrambledSobol64)+(VECTOR_SIZE +1)* 
            sizeof(long long int))*streamNum  //init_randstate
            + streamNum* max_stateNum* sizeof(float) * sz_burst * reSampleTimes //set_params_buff
            +streamNum * reSampleTimes * sizeof(retype)*streamNum //setBurstBd
            + max_stateNum *(3+2*max_stateNum*max_stateNum)* sizeof(float)*streamNum ; //set_nstates
        size_t free, total;
        hipMemGetInfo( &free, &total );
        size_t M200=(300<<20);
        if((free-reserved_gpumem)<=0)
            return false;
        size_t tot_conversion_buff_sz = free-reserved_gpumem-M200>0 ? free-reserved_gpumem-M200 : (size_t)((free-reserved_gpumem)*.90); 
        // tot_conversion_buff_sz/=streamNum;
        printf("tot_conversion_buff_sz: %zu\n",tot_conversion_buff_sz);
        int *tcc=new int[streamNum];

        for (int sid=0;sid<streamNum;sid++){
            // printf("n:%d\n",(end_burst[sid] - begin_burst[sid]));
            // printf("N:%f,streamNum=%d\n",N,streamNum);
            // printf("d:%f\n",(double)tot_conversion_buff_sz);
            // conversion_buff_sz[sid]=(int)floor(((double)tot_conversion_buff_sz)*(end_burst[sid] - begin_burst[sid])/N);
            conversion_buff_sz[sid]=(int)floor(((double)tot_conversion_buff_sz)/streamNum);
            tcc[sid]=(int)(conversion_buff_sz[sid]/(sz_burst*sizeof(int)));
            if(sid==0){
                printf("conversion_buff_sz[%d] %d\n",sid,conversion_buff_sz[sid]);
                printf("max_conversion_capacity[%d]: %d\n",sid,tcc[sid]);
            }
            g_conversion_buff[sid]=(int *)(mr->malloc(conversion_buff_sz[sid],streams[sid]));
        }
        // CUDA_CHECK_RETURN(hipMemcpy(g_conversion_capacity, tcc, streamNum*sizeof(int), hipMemcpyHostToDevice));
        delete[] tcc;
        data_gpu_inited=false;
        return true;
    }
    else
        return false;
}
void mc::init_randstate(int N, int sid) {
    int NN;
    int oldNN = oldN[sid] * reSampleTimes;
    if (N == -1) {
        NN = sz_burst * reSampleTimes;
    } else {
        NN = N * reSampleTimes;
    }
    gridSize[sid] = (NN + blockSize - 1) / blockSize;
    // for (int sid=0;sid<streamNum;sid++){

    // hipFree ( devStates[sid]) returned an illegal memory access was
    // encountered(700) at /home/liuk/data/proj/gSMFRETda/src/mc.cu:303
    // CUDA_CHECK_RETURN(hipFree ( devStates[sid]));
    // CUDA_CHECK_RETURN(hipFree ( devQStates[sid]));

    // printf("%d init_randstate, oldNN=%ld, devStates[sid]=%p\n", sid,
        //    oldNN * sizeof(rk_state), devStates[sid]);
    mr->free(devStates[sid], oldNN * sizeof(rk_state), streams[sid]);

    mr->free(devQStates[sid], oldNN * sizeof(hiprandStateScrambledSobol64),
             streams[sid]);
    devStates[sid] = (rk_state *)(mr->malloc(NN * sizeof(rk_state), streams[sid]));
    devQStates[sid] = (hiprandStateScrambledSobol64 *)(mr->malloc(
        NN * sizeof(hiprandStateScrambledSobol64), streams[sid]));
    // CUDA_CHECK_RETURN(hipMalloc ( (void **)&(devStates[sid]), NN*sizeof
    // (rk_state ) )); CUDA_CHECK_RETURN(hipMalloc ( (void
    // **)&(devQStates[sid]), NN*sizeof( hiprandStateScrambledSobol64)  ));

    // CUDA_CHECK_RETURN(hipFree (devDirectionVectors64[sid]));
    // CUDA_CHECK_RETURN(hipFree (devScrambleConstants64[sid]));
    mr->free(devDirectionVectors64[sid], oldNN * sizeof(hiprandDirectionVectors64_t),
             streams[sid]);
    mr->free(devScrambleConstants64[sid], oldNN * sizeof(unsigned long long int),
             streams[sid]);
    devDirectionVectors64[sid] = (unsigned long long int *)(mr->malloc(
        NN * VECTOR_SIZE * sizeof(long long int), streams[sid]));
    devScrambleConstants64[sid] = (unsigned long long int *)(mr->malloc(
        NN * sizeof(long long int), streams[sid]));
    // CUDA_CHECK_RETURN(hipMalloc((void **)&(devDirectionVectors64[sid]),
    // NN * VECTOR_SIZE * sizeof(long long int)));
    // CUDA_CHECK_RETURN(hipMalloc((void **)&(devScrambleConstants64[sid]),
    // NN * sizeof(long long int) ));
    /*
    hiprandStatus_t curandResult =hiprandGetDirectionVectors64(
    &(hostVectors64[sid]), HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6); if
    (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not get direction vectors for quasi-random number
    generator: "); msg += curandResult; throw std::runtime_error(msg);
    }
    curandResult=hiprandGetScrambleConstants64( &(hostScrambleConstants64[sid]));
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not get direction vectors for quasi-random number
    generator: "); msg += curandResult; throw std::runtime_error(msg);
    }
    */
    // std::cout << sid << " devDirectionVectors64ing, size ="
    //           << NN * VECTOR_SIZE * sizeof(long long int) << std::endl;
    int n = 0;
    int tNN = NN;
    while (tNN > 0) {
        int size = (tNN > 20000) ? 20000 : tNN;
        unsigned long long int *buf = devScrambleConstants64[sid];
        CUDA_CHECK_RETURN(hipMemcpyAsync(buf + n * 20000, hostScrambleConstants64,
                                          size * sizeof(unsigned long long int),
                                          hipMemcpyHostToDevice, streams[sid]));
        // std::cout << "n = " << n << ", size = " << size << std::endl;
        buf = devDirectionVectors64[sid];
        CUDA_CHECK_RETURN(
            hipMemcpyAsync(buf + n * 20000 * sizeof(hiprandDirectionVectors64_t) /
                                      sizeof(unsigned long long int),
                            hostVectors64, size * sizeof(hiprandDirectionVectors64_t),
                            hipMemcpyHostToDevice, streams[sid]));

        tNN -= size;
        n++;
    }
    // std::cout << sid << " devDirectionVectors64ed \n";

    setup_kernel<<<gridSize[sid], blockSize, 0, streams[sid]>>>(
        devStates[sid], 0, /*time(NULL)*/ NN, devDirectionVectors64[sid],
        devScrambleConstants64[sid], devQStates[sid]);
    // CUDAstream_CHECK_LAST_ERROR;
    CUDA_CHECK_RETURN(hipStreamSynchronize(streams[sid]));
    // }
    oldN[sid] = N;
}

void mc::init_data_gpu(vector<int64_t> &istart, vector<int64_t> &start,
                       vector<int64_t> &stop, std::vector<int> &phCount,
                       long _sz_tag, int64_t *burst_ad, int64_t *burst_dd,
                       vector<float> &T_burst_duration, vector<float> &SgDivSr,
                       float &iclk_p, float &ibg_ad_rate, float &ibg_dd_rate) {
    clk_p = iclk_p;
    bg_ad_rate = ibg_ad_rate;
    bg_dd_rate = ibg_dd_rate;
    sz_tag = _sz_tag;
    sz_burst = start.size();
    int sidx = 0;
    g_phCount =
        (int *)mr->malloc(sizeof(int) * sz_burst, streams[(sidx) % streamNum]);
    g_conversion_capacity = (int *)(mr->malloc(sizeof(int)*streamNum, streams[(sidx++) % streamNum]));
    CUDA_CHECK_RETURN(hipMemcpyAsync(g_phCount, phCount.data(),
                                      sizeof(int) * sz_burst, hipMemcpyHostToDevice,
                                      streams[(sidx++) % streamNum]));
    g_burst_ad =
        (int64_t *)mr->malloc(sizeof(int64_t) * sz_tag, streams[(sidx) % streamNum]);
    CUDA_CHECK_RETURN(hipMemcpyAsync(g_burst_ad, burst_ad, sizeof(int64_t) * sz_tag,
                                      hipMemcpyHostToDevice,
                                      streams[(sidx++) % streamNum]));
    g_burst_dd =
        (int64_t *)mr->malloc(sizeof(int64_t) * sz_tag, streams[(sidx) % streamNum]);
    CUDA_CHECK_RETURN(hipMemcpyAsync(g_burst_dd, burst_dd, sizeof(int64_t) * sz_tag,
                                      hipMemcpyHostToDevice,
                                      streams[(sidx++) % streamNum]));

    g_istart = (int64_t *)mr->malloc(sizeof(int64_t) * sz_burst,
                                     streams[(sidx) % streamNum]);
    CUDA_CHECK_RETURN(
        hipMemcpyAsync(g_istart, istart.data(), sizeof(int64_t) * sz_burst,
                        hipMemcpyHostToDevice, streams[(sidx++) % streamNum]));
    // CUDA_CHECK_RETURN(hipMalloc((void **)&g_start,
    // sizeof(int64_t)*sz_burst));
    g_start = (int64_t *)mr->malloc(sizeof(int64_t) * sz_burst,
                                    streams[(sidx) % streamNum]);
    CUDA_CHECK_RETURN(
        hipMemcpyAsync(g_start, start.data(), sizeof(int64_t) * sz_burst,
                        hipMemcpyHostToDevice, streams[(sidx++) % streamNum]));
    // CUDA_CHECK_RETURN(hipMalloc((void **)&g_stop,
    // sizeof(int64_t)*sz_burst));
    g_stop = (int64_t *)mr->malloc(sizeof(int64_t) * sz_burst,
                                   streams[(sidx) % streamNum]);
    CUDA_CHECK_RETURN(
        hipMemcpyAsync(g_stop, stop.data(), sizeof(int64_t) * sz_burst,
                        hipMemcpyHostToDevice, streams[(sidx++) % streamNum]));

    // CUDA_CHECK_RETURN(hipMalloc((void **)&g_burst_duration,
    // sizeof(float)*sz_burst));
    g_burst_duration =
        (float *)mr->malloc(sizeof(float) * sz_burst, streams[(sidx) % streamNum]);
    CUDA_CHECK_RETURN(hipMemcpyAsync(
        g_burst_duration, T_burst_duration.data(), sizeof(float) * sz_burst,
        hipMemcpyHostToDevice, streams[(sidx++) % streamNum]));
    // CUDA_CHECK_RETURN(hipMalloc((void **)&g_SgDivSr,
    // sizeof(float)*sz_burst));
    g_SgDivSr =
        (float *)mr->malloc(sizeof(float) * sz_burst, streams[(sidx) % streamNum]);
    CUDA_CHECK_RETURN(
        hipMemcpyAsync(g_SgDivSr, SgDivSr.data(), sizeof(float) * sz_burst,
                        hipMemcpyHostToDevice, streams[(sidx++) % streamNum]));
    // CUDA_CHECK_RETURN(hipMalloc((void **)&gchi2, sizeof(float)));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    data_gpu_inited=true;
}

int mc::setBurstBd(int cstart, int cstop, int sid) {
    int rcstart = cstart;
    int rcstop = cstop;
    if (cstop >= sz_burst || cstop < 0)
        rcstop = sz_burst;
    if (cstart >= rcstop) {
        rcstart = 0;
    }
    int N = rcstop - rcstart;
    if (cstart == -1) {
        gridSize[sid] = -blockSize;
        N = -1;
    }
    if (end_burst[sid] - begin_burst[sid] != N) {
        begin_burst[sid] = rcstart;
        end_burst[sid] = rcstop;
        // int dimension=256;
        // dim3 threads = dim3(dimension, 1);
        // int blocksCount = ceil(float(N)/dimension);
        // dim3 blocks  = dim3(blocksCount, 1);
        gridSize[sid] = (N* reSampleTimes + blockSize - 1) / blockSize;
        // if (debug)
        //     cout << gridSize[sid] << " g " << N * reSampleTimes << " tN "
        //          << blockSize << " bS " << mcE[sid] << endl;

        // CUDA_CHECK_RETURN(hipFree((void*)mcE[sid]));
        mr->free(mcE[sid], oldN[sid] * reSampleTimes * sizeof(retype), streams[sid]);
        // if (debug)
        //     cout << "mcE[sid]" << sid << ":" << mcE[sid] << endl;
        // CUDA_CHECK_RETURN(hipMalloc((void **)&(mcE[sid]), N *reSampleTimes*
        // sizeof(retype)));
        mcE[sid] =
            (retype *)(mr->malloc(N * reSampleTimes * sizeof(retype), streams[sid]));

        // if (debug)
        //     cout << "mcE[sid]" << sid << ":" << mcE[sid] << endl;
        CUDA_CHECK_RETURN(hipHostFree((void *)hmcE[sid]));
        CUDA_CHECK_RETURN(hipHostAlloc((void **)&(hmcE[sid]),
                                        N * reSampleTimes * sizeof(retype),
                                        hipHostMallocDefault));
    }
    return N;
}

void mc::run_kernel(int N, int sid) {
    AtomicWriter(debug, debugLevel::gpu)
        << "mcE[sid]" << sid << ":" << mcE[sid] << "\n";

    mc_kernel<<<gridSize[sid], blockSize, 0, streams[sid]>>>(
        g_start, g_stop, g_burst_ad, g_burst_dd, g_istart, g_phCount,
        g_burst_duration, /*g_SgDivSr,*/
        clk_p, bg_ad_rate, bg_dd_rate, gpe[sid], gpv[sid], gpk[sid], gpp_i2j[sid], gpp[sid],
        g_P_i2j[sid], N, s_n[sid], devQStates[sid], devStates[sid], mcE[sid],
        reSampleTimes, debug & debugLevel::kernel);
    // CUDAstream_CHECK_LAST_ERROR;
    // CUDA_CHECK_RETURN(hipStreamSynchronize(streams[sid]));
    if (debug & debugLevel::gpu) {
        cout << "sid:" << sid << endl;
        cout << "streams[" << sid << "]:" << streams[sid] << endl;
        cout << "hmcE[sid]" << sid << ":" << hmcE[sid] << endl;
    }
    CUDA_CHECK_RETURN(hipMemcpyAsync(hmcE[sid], mcE[sid],
                                      N * reSampleTimes * sizeof(retype),
                                      hipMemcpyDeviceToHost, streams[sid]));
}
/**
 * @brief  Query if the stream finished
 *
 * @param sid
 * @return true
 * @return false
 */
bool mc::streamQuery(int sid) {
    if (sid < 0 || sid >= streamNum)
        return false;
    if (hipStreamQuery(streams[sid]) == hipSuccess)
        return true;
    return false;
}
void mc::get_res(int sid, int N) {
    // CUDA_CHECK_RETURN(hipStreamSynchronize(streams[sid]));
    if (debug) {
        std::vector<retype> my_vector(hmcE[sid], hmcE[sid] + N * reSampleTimes);
        auto maxPosition = max_element(std::begin(my_vector), std::end(my_vector));
        for (int ip = 0; ip < 10; ip++)
            printf("%2.4f \t", *(hmcE[sid] + ip));
        cout << endl << sid << "======" << my_vector.size() << "\n";
        cout << *maxPosition << ","
             << accumulate(my_vector.begin(), my_vector.end(), 0.0) /
                    my_vector.size()
             << endl;
        // savehdf5("r.hdf5", "/r",my_vector);
    }
}

mc::~mc() {
    free_data_gpu();
    free (matK);
    free(matP_i2j);
    free (matP);
    delete (mr);
    for (int sid = 0; sid < streamNum; sid++) {
        CUDA_CHECK_RETURN(hipStreamSynchronize(streams[sid]));
        CUDA_CHECK_RETURN(hipStreamDestroy(streams[sid]));
    }
    free(streams);
    delete[](s_n);
    delete[](conversion_buff_sz);
    delete[](oldN);
    delete[](begin_burst);
    delete[](end_burst);
    delete[](gridSize);
    free(hmcE);
    free(mcE);
    // free(hpe);
    // free(hpv);
    // free(hpp);
    // free(hpk);
    free(g_conversion_buff);
    free(gpe);
    free(gpv);
    free(gpp);
    free(gpk);
    free(gpp_i2j);
    free(g_P_i2j);
    // free(hostVectors64);
    // free(hostScrambleConstants64);

    free(devStates);
    free(devQStates);
    free(devDirectionVectors64);
    free(devScrambleConstants64);
    if (profiler) {
        CUDA_CHECK_RETURN(hipProfilerStop());
        std::cout << "hipProfilerStop" << std::endl;
    }
    hipDeviceReset();
}

void mc::free_data_gpu() {
    // hipDeviceSynchronize();

    // // CUDA_CHECK_RETURN(hipFree(g_mask_ad));
    // // CUDA_CHECK_RETURN(hipFree(g_mask_dd));
    // CUDA_CHECK_RETURN(hipFree(g_start));
    // CUDA_CHECK_RETURN(hipFree(g_stop));
    // // CUDA_CHECK_RETURN(hipFree(g_istart));
    // // CUDA_CHECK_RETURN(hipFree(g_istop));
    // // CUDA_CHECK_RETURN(hipFree(g_times_ms));
    // CUDA_CHECK_RETURN(hipFree(g_SgDivSr));
    // CUDA_CHECK_RETURN(hipFree(g_burst_duration));
    int sidx = 0;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    mr->free(g_conversion_capacity, sizeof(int) * streamNum, streams[(sidx++) % streamNum]);
    mr->free(g_phCount, sizeof(int) * sz_burst, streams[(sidx++) % streamNum]);
    mr->free(g_burst_ad, sizeof(int64_t) * sz_tag, streams[(sidx++) % streamNum]);
    mr->free(g_burst_dd, sizeof(int64_t) * sz_tag, streams[(sidx++) % streamNum]);
    mr->free(g_start, sizeof(int64_t) * sz_burst, streams[(sidx++) % streamNum]);
    mr->free(g_istart, sizeof(int64_t) * sz_burst, streams[(sidx++) % streamNum]);
    mr->free(g_stop, sizeof(int64_t) * sz_burst, streams[(sidx++) % streamNum]);
    mr->free(g_burst_duration, sizeof(float) * sz_burst,
             streams[(sidx++) % streamNum]);
    mr->free(g_SgDivSr, sizeof(float) * sz_burst, streams[(sidx++) % streamNum]);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    for (int sid = 0; sid < streamNum; sid++) {
       delete(matK[sid]);
       delete(matP_i2j[sid]);
       delete(matP[sid]);
        // hipStreamSynchronize(streams[sid]);
        // CUDA_CHECK_RETURN(hipFree(gpe[sid]));
        // CUDA_CHECK_RETURN(hipFree(gpv[sid]));
        // CUDA_CHECK_RETURN(hipFree(gpp[sid]));
        // CUDA_CHECK_RETURN(hipFree(gpk[sid]));
        mr->free(gpe[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpv[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpp[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpk[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpp_i2j[sid], s_n[sid] * sizeof(float), streams[sid]);
        // CUDA_CHECK_RETURN(hipFree(mcE[sid]));
        int oldNN = oldN[sid] * reSampleTimes;
        mr->free(mcE[sid], oldNN * sizeof(retype), streams[sid]);
        mr->free(g_P_i2j[sid], s_n[sid] * sizeof(float) * oldNN, streams[sid]);
        // CUDA_CHECK_RETURN(hipFree(devStates[sid]));
        // CUDA_CHECK_RETURN(hipFree(devQStates[sid]));
        // CUDA_CHECK_RETURN(hipFree(devDirectionVectors64[sid]));
        // CUDA_CHECK_RETURN(hipFree(devScrambleConstants64[sid]));
        mr->free(devStates[sid], oldNN * sizeof(rk_state), streams[sid]);
        mr->free(devQStates[sid], oldNN * sizeof(hiprandStateScrambledSobol64),
                 streams[sid]);
        mr->free(devDirectionVectors64[sid],
                 oldNN * VECTOR_SIZE * sizeof(long long int), streams[sid]);
        mr->free(devScrambleConstants64[sid], oldNN * sizeof(long long int),
                 streams[sid]);

        cout << "free sid" << sid << endl;
        CUDA_CHECK_RETURN(hipHostFree(hmcE[sid]));
    }
}

int mc::set_nstates(int n, int sid) {
    int r = n;
    if (s_n[sid] != n) {
        r = s_n[sid];
        CUDA_CHECK_RETURN(hipStreamSynchronize(streams[sid]));
        // std::cout << "CUDA_CHECK_RETURN( hipStreamSynchronize(streams[" << sid
        //           << "])\n";
        // CUDA_CHECK_RETURN(hipHostFree(hpe[sid]));
        // CUDA_CHECK_RETURN(hipHostFree(hpv[sid]));
        // CUDA_CHECK_RETURN(hipHostFree(hpp[sid]));
        // CUDA_CHECK_RETURN(hipHostFree(hpk[sid]));
        // // CUDA_CHECK_RETURN(hipFree(gpe[sid]));
        // // CUDA_CHECK_RETURN(hipFree(gpv[sid]));
        // // CUDA_CHECK_RETURN(hipFree(gpp[sid]));
        // // CUDA_CHECK_RETURN(hipFree(gpk[sid]));
        // CUDA_CHECK_RETURN(hipHostAlloc((void **)&(hpe[sid]),
        // n*sizeof(float),hipHostMallocDefault));
        // CUDA_CHECK_RETURN(hipHostAlloc((void **)&(hpv[sid]),
        // n*sizeof(float),hipHostMallocDefault));
        // CUDA_CHECK_RETURN(hipHostAlloc((void **)&(hpp[sid]),
        // n*sizeof(float),hipHostMallocDefault));
        // CUDA_CHECK_RETURN(hipHostAlloc((void **)&(hpk[sid]),
        // n*n*sizeof(float),hipHostMallocDefault)); if (gpe[sid]!=0)
        // CUDA_CHECK_RETURN(hipFree(gpe[sid])); if (gpv[sid]!=0)
        // CUDA_CHECK_RETURN(hipFree(gpv[sid])); if (gpp[sid]!=0)
        // CUDA_CHECK_RETURN(hipFree(gpp[sid])); if (gpk[sid]!=0)
        // CUDA_CHECK_RETURN(hipFree(gpk[sid]));
        mr->free(gpe[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpv[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpp[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpk[sid], s_n[sid] * sizeof(float), streams[sid]);
        mr->free(gpp_i2j[sid], s_n[sid] * sizeof(float), streams[sid]);
        delete (matK[sid]);
        matK[sid]=new float[n*n];
        delete (matP_i2j[sid]);
        matP_i2j[sid] = new float[n * n];
        delete(matP[sid]);
        matP[sid]=new float[n];        
        gpe[sid] = (float *)(mr->malloc(n * sizeof(float), streams[sid]));
        gpv[sid] = (float *)(mr->malloc(n * sizeof(float), streams[sid]));
        gpp[sid] = (float *)(mr->malloc(n * sizeof(float), streams[sid]));
        gpk[sid] = (float *)(mr->malloc(n * n* sizeof(float), streams[sid]));
        gpp_i2j[sid] = (float*)(mr->malloc(n * n * sizeof(float), streams[sid]));
        // CUDA_CHECK_RETURN(hipMalloc((void **)&(gpe[sid]), n*sizeof(float)));
        // CUDA_CHECK_RETURN(hipMalloc((void **)&(gpv[sid]), n*sizeof(float)));
        // CUDA_CHECK_RETURN(hipMalloc((void **)&(gpp[sid]), n*sizeof(float)));
        // CUDA_CHECK_RETURN(hipMalloc((void **)&(gpk[sid]),
        // n*n*sizeof(float)));

        // CUDA_CHECK_RETURN(_rmmReAlloc((void **)&(gpe[sid]),
        // n*sizeof(float),streams[sid])); CUDA_CHECK_RETURN(_rmmReAlloc((void
        // **)&(gpv[sid]), n*sizeof(float),streams[sid]));
        // CUDA_CHECK_RETURN(_rmmReAlloc((void **)&(gpp[sid]),
        // n*sizeof(float),streams[sid])); CUDA_CHECK_RETURN(_rmmReAlloc((void
        // **)&(gpk[sid]), n*n*sizeof(float),streams[sid]));
        s_n[sid] = n;
    }
    return r;
}

void mc::set_params_buff(int oldS_n, int N_sid, int sid) {
    mr->free(g_P_i2j[sid], oldS_n * sizeof(float) * oldN[sid] * reSampleTimes,
             streams[sid]);
    g_P_i2j[sid] = (float *)mr->malloc(
        s_n[sid] * sizeof(float) * N_sid * reSampleTimes, streams[sid]);
}

bool mc::set_params(int n, int sid, vector<float> &args) {
    bool r;
    vecFloatMapper evargs(args.data(), n * n + n);
    // RowVectorXf eargs=evargs(seqN(0,n));
    RowVectorXf eargs = evargs.block(0, 0, 1, n);
    float *peargs = eargs.data();
    // RowVectorXf kargs=evargs(seqN(n,n*n-n));
    RowVectorXf kargs = evargs.block(0, n, 1, n * n - n);
    // RowVectorXf vargs=evargs(seqN(n*n,n));
    RowVectorXf vargs = evargs.block(0, n * n, 1, n);
    float *pvargs = vargs.data();

    r = genMatK(matK[sid], n, kargs);
    genP_i2j(matK[sid], matP_i2j[sid], n);
    //&matK不可修改，但是matK的值可以修改
    r = r && genMatP(matP[sid], matK[sid],n);
    // cout<<"[K]:\n"<<*matK<<endl;
    // memcpy(hpp[sid], matP->data(), sizeof(float)*n);
    // todo
    CUDA_CHECK_RETURN(hipMemcpyAsync(gpe[sid], peargs, sizeof(float) * n,
                                      hipMemcpyHostToDevice, streams[sid]));
    CUDA_CHECK_RETURN(hipMemcpyAsync(gpv[sid], pvargs, sizeof(float) * n,
                                      hipMemcpyHostToDevice, streams[sid]));
    //std::cout << *(matK[sid]) << std::endl;
    // for (int di=0;di<n;di++){
    //     std::cout << *(matP[sid]+di) << std::endl;
    //     for (int dj=0;dj<n;dj++)
    //         std::cout << *(matK[sid]+di+dj*n) << "\t";
    //     std::cout<< std::endl;
    // }
    CUDA_CHECK_RETURN(hipMemcpyAsync(gpk[sid], matK[sid], sizeof(float) * n * n,
                                     hipMemcpyHostToDevice, streams[sid]));
    CUDA_CHECK_RETURN(hipMemcpyAsync(gpp_i2j[sid], matP_i2j[sid], sizeof(float) * n * n,
        hipMemcpyHostToDevice, streams[sid]));
    CUDA_CHECK_RETURN(hipMemcpyAsync(gpp[sid], matP[sid], sizeof(float) * n,
                                     hipMemcpyHostToDevice, streams[sid]));

    CUDA_CHECK_RETURN(hipStreamSynchronize(streams[sid]));
    return r;
}
