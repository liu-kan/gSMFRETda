#include "hip/hip_runtime.h"
#include "mc.hpp"
#include "cuda_tools.hpp"
#include "loadHdf5.hpp"
#include <time.h>
#define VECTOR_SIZE 64

#include "binom.cuh"
#include "gen_rand.cuh"


__global__ void mc_kernel(float *chi2, int64_t* start,int64_t* stop,
    uint32_t* istart,uint32_t* istop,
    int64_t* times_ms,
    unsigned char* mask_ad,unsigned char* mask_dd,
    float* T_burst_duration,float* SgDivSr,
    float clk_p,float bg_ad_rate,float bg_dd_rate,long sz_tag,int sz_burst ,
    float* gpe,float* gpv,float* gpk,float* gpp,
    int N,int n,hiprandStateScrambledSobol64 *devQStates,rk_state *devStates, retype *intr){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N){
        arrUcharMapper mask_adA(mask_ad,sz_tag);
        // intr[idx]=drawDisIdx(n,gpp,devQStates+idx);
        // intr[idx]=drawTau(25,devQStates+idx);
        // float t=1;
        // draw_P_B_Tr(intr+idx,35,1,&t,6 ,devQStates+idx);
        // intr[idx]=drawE(3.0,6,devQStates+idx);        
        intr[idx]=drawA_fi_e(devStates+idx, 5, 0.7) ;
    }
    
}

__global__ void setup_kernel  (rk_state * state, unsigned long seed , int N,
    unsigned long long * sobolDirectionVectors, 
    unsigned long long *sobolScrambleConstants, 
    hiprandStateScrambledSobol64* stateQ)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N){
        // hiprand_init ( seed, idx, 0, &state[idx] );        
        hiprand_init(sobolDirectionVectors + VECTOR_SIZE*idx, 
            sobolScrambleConstants[idx], 
            1234, 
            &stateQ[idx]);
        unsigned long long llseed=hiprand(stateQ+idx);    
        rk_seed(llseed,state+idx);
    }
} 

mc::mc(int id){
    devid=id;
    matK=NULL;matP=NULL;
    hpe=hpv=hpk=hpp=gpe=gpv=gpp=gpk=NULL;    
    devStates=NULL;
    devQStates=NULL;        
    CUDA_CHECK_RETURN(hipSetDevice(devid));
    hostVectors64=NULL;
    hostScrambleConstants64=NULL;
    devDirectionVectors64=NULL;
    devScrambleConstants64=NULL;
    
}

void mc::init_data_gpu(vector<int64_t>& start,vector<int64_t>& stop,
        vector<uint32_t>& istart,vector<uint32_t>& istop,
        vector<int64_t>& times_ms,
        vector<unsigned char>& mask_ad,vector<unsigned char>& mask_dd,
        vector<float>& T_burst_duration,vector<float>& SgDivSr,
        float& iclk_p,float& ibg_ad_rate,float& ibg_dd_rate){    
    clk_p=iclk_p;bg_ad_rate=ibg_ad_rate;bg_dd_rate=ibg_dd_rate;    
    sz_tag=mask_ad.size();                    
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hchi2, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_mask_ad, sizeof(unsigned char)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_mask_ad, mask_ad.data(), sizeof(unsigned char)*sz_tag, 
        hipMemcpyHostToDevice));    
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_mask_dd, sizeof(unsigned char)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_mask_dd, mask_dd.data(), sizeof(unsigned char)*sz_tag,hipMemcpyHostToDevice));
    sz_burst=start.size();         
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_start, sizeof(int64_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_start, start.data(), sizeof(int64_t)*sz_burst,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_stop, sizeof(int64_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_stop, stop.data(), sizeof(int64_t)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_istart, sizeof(uint32_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_istart, istart.data(), sizeof(uint32_t)*sz_burst,hipMemcpyHostToDevice));    
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_istop, sizeof(uint32_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_istop, istop.data(), sizeof(uint32_t)*sz_burst,hipMemcpyHostToDevice));            
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_times_ms, sizeof(int64_t)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_times_ms, times_ms.data(), sizeof(int64_t)*sz_tag,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_burst_duration, sizeof(float)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_burst_duration, T_burst_duration.data(), sizeof(float)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_SgDivSr, sizeof(float)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_SgDivSr, SgDivSr.data(), sizeof(float)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&gchi2, sizeof(float)));
}

void mc::run_kernel(int cstart,int cstop){  
    int N=cstop-cstart;
    int dimension=128;  
    dim3 threads = dim3(dimension, 1);
    int blocksCount = ceil(N / dimension);
    dim3 blocks  = dim3(blocksCount, 1);    
    CUDA_CHECK_RETURN(hipFree ( devStates));
    CUDA_CHECK_RETURN(hipFree ( devQStates));    
    CUDA_CHECK_RETURN(hipMalloc ( (void **)&devStates, N*sizeof (rk_state ) ));
    CUDA_CHECK_RETURN(hipMalloc ( (void **)&devQStates, N*sizeof( hiprandStateScrambledSobol64) ));    
    CUDA_CHECK_RETURN(hipFree (devDirectionVectors64));
    CUDA_CHECK_RETURN(hipFree (devScrambleConstants64));
    // CUDA_CHECK_RETURN
    (hiprandGetDirectionVectors64( &hostVectors64, 
        HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6));
    // CUDA_CHECK_RETURN
    (hiprandGetScrambleConstants64( &hostScrambleConstants64));              
    CUDA_CHECK_RETURN(hipMalloc((void **)&(devDirectionVectors64), 
    N * VECTOR_SIZE * sizeof(long long int)));        
    CUDA_CHECK_RETURN(hipMemcpy(devDirectionVectors64, hostVectors64,
    N * VECTOR_SIZE * sizeof(long long int), 
    hipMemcpyHostToDevice)); 
    CUDA_CHECK_RETURN(hipMalloc((void **)&(devScrambleConstants64), 
    N * sizeof(long long int)));
    CUDA_CHECK_RETURN(hipMemcpy(devScrambleConstants64, hostScrambleConstants64,
    N * sizeof(long long int), 
    hipMemcpyHostToDevice));
    setup_kernel <<<blocks, threads>>>(devStates, 0,/*time(NULL)*/ N ,
        devDirectionVectors64, devScrambleConstants64, devQStates);

    retype *intr,*hintr;
    CUDA_CHECK_RETURN(hipMalloc((void **)&intr, N * sizeof(retype)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hintr, N * sizeof(retype)));
    mc_kernel<<<blocks, threads>>>(gchi2, g_start,g_stop,
        g_istart,g_istop,
        g_times_ms,
        g_mask_ad,g_mask_dd,
        g_burst_duration,g_SgDivSr,
        clk_p,bg_ad_rate,bg_dd_rate,sz_tag,sz_burst ,
        gpe,gpv,gpk,gpp,N,s_n,devQStates,devStates, intr);
    CUDA_CHECK_RETURN(hipMemcpy(hintr, intr,N * sizeof(retype), hipMemcpyDeviceToHost));        
    std::vector<retype> my_vector(hintr, hintr + N);
    for (int ip=0;ip<N;ip++)
        cout<<my_vector.at(ip)<<" ";
    cout<<endl;
    savehdf5("r.hdf5", "/r",my_vector);
    CUDA_CHECK_RETURN(hipFree(intr));
    CUDA_CHECK_RETURN(hipHostFree(hintr));
}

mc::~mc(){
    free_data_gpu();
    delete(matK);delete(matP);
}

void mc::free_data_gpu(){            
    CUDA_CHECK_RETURN(hipFree(g_mask_ad));
    CUDA_CHECK_RETURN(hipFree(g_mask_dd));
    CUDA_CHECK_RETURN(hipFree(g_start));
    CUDA_CHECK_RETURN(hipFree(g_stop));    
    CUDA_CHECK_RETURN(hipFree(g_istart));
    CUDA_CHECK_RETURN(hipFree(g_istop)); 
    CUDA_CHECK_RETURN(hipFree(g_times_ms));
    CUDA_CHECK_RETURN(hipFree(g_SgDivSr));
    CUDA_CHECK_RETURN(hipFree(g_burst_duration));

    // CUDA_CHECK_RETURN(hipFree(r_size));
    hipDeviceSynchronize();
    cout<<"rsize:"<<*hchi2<<endl;
    CUDA_CHECK_RETURN(hipHostFree(hchi2));
    CUDA_CHECK_RETURN(hipHostFree(hpe));
    CUDA_CHECK_RETURN(hipHostFree(hpv));
    CUDA_CHECK_RETURN(hipHostFree(hpp));
    CUDA_CHECK_RETURN(hipHostFree(hpk));
    CUDA_CHECK_RETURN(hipFree(gpe));
    CUDA_CHECK_RETURN(hipFree(gpv));
    CUDA_CHECK_RETURN(hipFree(gpp));
    CUDA_CHECK_RETURN(hipFree(gpk));        
}

bool mc::set_nstates(int n){
    s_n=n;
    bool r;
    CUDA_CHECK_RETURN(hipHostFree(hpe));
    CUDA_CHECK_RETURN(hipHostFree(hpv));
    CUDA_CHECK_RETURN(hipHostFree(hpp));
    CUDA_CHECK_RETURN(hipHostFree(hpk));
    CUDA_CHECK_RETURN(hipFree(gpe));
    CUDA_CHECK_RETURN(hipFree(gpv));
    CUDA_CHECK_RETURN(hipFree(gpp));
    CUDA_CHECK_RETURN(hipFree(gpk));    
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpe, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpv, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpp, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpk, n*n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpe, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpv, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpp, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpk, n*n*sizeof(float)));    
    return r;
}

bool mc::set_params(vector<float>& args){
    int n=s_n;
    vecFloatMapper evargs(args.data(),n*n+n);    
    cout<<evargs<<endl;
    eargs=evargs(seqN(0,n));
    float *peargs=eargs.data();
    kargs=evargs(seqN(n,n*n-n));    
    vargs=evargs(seqN(n*n,n));    
    float *pvargs=vargs.data();
    bool r=genMatK(&matK,n,kargs);
    //&matK不可修改，但是matK的值可以修改    
    r=r&&genMatP(&matP,matK);    
    cout<<"p:"<<*matP<<endl;
    memcpy(hpe, peargs, sizeof(float)*n);
    memcpy(hpv, pvargs, sizeof(float)*n);
    memcpy(hpk, matK->data(), sizeof(float)*n*n);
    memcpy(hpp, matP->data(), sizeof(float)*n);
    CUDA_CHECK_RETURN(hipMemcpy(gpe,hpe,sizeof(float)*n,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpv,hpv, sizeof(float)*n,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpk,hpk, sizeof(float)*n*n,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpp,hpp, sizeof(float)*n,hipMemcpyHostToDevice));    
    return r;
}