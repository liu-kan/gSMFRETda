#include "hip/hip_runtime.h"
#include "mc.hpp"

#include "cuda_tools.hpp"


__global__ void mc_kernel(float *chi2, int64_t* start,int64_t* stop,
    uint32_t* istart,uint32_t* istop,
    int64_t* times_ms,
    unsigned char* mask_ad,unsigned char* mask_dd,
    float* T_burst_duration,float* SgDivSr,
    float clk_p,float bg_ad_rate,float bg_dd_rate,long sz_tag,int sz_burst ){
    arrUcharMapper mask_adA(mask_ad,sz_tag);
    // *rsize=mask_adA.cols();
}

mc::mc(int id){
    devid=id;
    matK=NULL;matP=NULL;
}

void mc::init_data_gpu(vector<int64_t>& start,vector<int64_t>& stop,
        vector<uint32_t>& istart,vector<uint32_t>& istop,
        vector<int64_t>& times_ms,
        vector<unsigned char>& mask_ad,vector<unsigned char>& mask_dd,
        vector<float>& T_burst_duration,vector<float>& SgDivSr,
        float& iclk_p,float& ibg_ad_rate,float& ibg_dd_rate){    
    clk_p=iclk_p;bg_ad_rate=ibg_ad_rate;bg_dd_rate=ibg_dd_rate;
    CUDA_CHECK_RETURN(hipSetDevice(devid));
    sz_tag=mask_ad.size();                
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hchi2, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_mask_ad, sizeof(unsigned char)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_mask_ad, mask_ad.data(), sizeof(unsigned char)*sz_tag, 
        hipMemcpyHostToDevice));    
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_mask_dd, sizeof(unsigned char)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_mask_dd, mask_dd.data(), sizeof(unsigned char)*sz_tag,hipMemcpyHostToDevice));
    sz_burst=start.size();         
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_start, sizeof(int64_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_start, start.data(), sizeof(int64_t)*sz_burst,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_stop, sizeof(int64_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_stop, stop.data(), sizeof(int64_t)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_istart, sizeof(uint32_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_istart, istart.data(), sizeof(uint32_t)*sz_burst,hipMemcpyHostToDevice));    
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_istop, sizeof(uint32_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_istop, istop.data(), sizeof(uint32_t)*sz_burst,hipMemcpyHostToDevice));            
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_times_ms, sizeof(int64_t)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_times_ms, times_ms.data(), sizeof(int64_t)*sz_tag,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_burst_duration, sizeof(float)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_burst_duration, T_burst_duration.data(), sizeof(float)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_SgDivSr, sizeof(float)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_SgDivSr, SgDivSr.data(), sizeof(float)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&gchi2, sizeof(float)));
}

void mc::run_kernel(){
    CUDA_CHECK_RETURN(hipSetDevice(devid));
    mc_kernel<<<1,1>>>(gchi2, g_start,g_stop,
        g_istart,g_istop,
        g_times_ms,
        g_mask_ad,g_mask_dd,
        g_burst_duration,g_SgDivSr,
        clk_p,bg_ad_rate,bg_dd_rate,sz_tag,sz_burst );
    CUDA_CHECK_RETURN(hipMemcpy(hchi2, gchi2, sizeof(float), hipMemcpyDeviceToHost));        
}

mc::~mc(){
    free_data_gpu();
    delete(matK);delete(matP);
}



void mc::free_data_gpu(){    
    CUDA_CHECK_RETURN(hipSetDevice(devid));
    CUDA_CHECK_RETURN(hipFree(g_mask_ad));
    CUDA_CHECK_RETURN(hipFree(g_mask_dd));
    CUDA_CHECK_RETURN(hipFree(g_start));
    CUDA_CHECK_RETURN(hipFree(g_stop));    
    CUDA_CHECK_RETURN(hipFree(g_istart));
    CUDA_CHECK_RETURN(hipFree(g_istop)); 
    CUDA_CHECK_RETURN(hipFree(g_times_ms));
    CUDA_CHECK_RETURN(hipFree(g_SgDivSr));
    CUDA_CHECK_RETURN(hipFree(g_burst_duration));
    // CUDA_CHECK_RETURN(hipFree(r_size));
    hipDeviceSynchronize();
    cout<<"rsize:"<<*hchi2<<endl;
    CUDA_CHECK_RETURN(hipHostFree(hchi2));
}
bool mc::set_params(int n,vector<float>& args){
    vecFloatMapper evargs(args.data(),n*n+n);    
    cout<<evargs<<endl;
    eargs=evargs(seqN(0,n));
    kargs=evargs(seqN(n,n*n-n));
    vargs=evargs(seqN(n*n,n));    
    bool r=genMatK(&matK,n,kargs);
    //&matK不可修改，但是matK的值可以修改    
    r=r&&genMatP(&matP,matK);
    // delete(matK);
    return r;
}