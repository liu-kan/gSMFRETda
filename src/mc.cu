#include "hip/hip_runtime.h"
#include "mc.hpp"
#include "cuda_tools.hpp"
#include "loadHdf5.hpp"
#include <time.h>
#define VECTOR_SIZE 64

#include "binom.cuh"
#include "gen_rand.cuh"
#include "cuList.cuh"

#define gamma 0.34
#define beta 1.42
#define DexDirAem 0.08
#define Dch2Ach 0.07

template <typename T>
__forceinline__ __device__ void binTimeHist(arrF* hist, arrI64& x,
         cuList<T> bins ){
    int binlen=bins.len;
    hist->resize(1,binlen-1);
    hist->setZero();
    int datalen=x.cols();
    for (int i=0;i<datalen;i++){
        if(x(i)==0)
            continue;
        int idxbin=1;
        do{
            T v=*(bins.at(idxbin));
            if (x(i)<v){
                ((*hist)(idxbin-1))+=1;
                break;
            }
            idxbin++;
        }while(idxbin<binlen);
    }
}
__global__ void mc_kernel(float *chi2, int64_t* start,int64_t* stop,
    uint32_t* istart,uint32_t* istop,
    int64_t* times_ms,
    unsigned char* mask_ad,unsigned char* mask_dd,
    float* T,float* SgDivSr,
    float clk_p,float bg_ad_rate,float bg_dd_rate,long sz_tag,int sz_burst ,
    float* gpe,float* gpv,float* gpk,float* gpp,
    int N,int s_n,hiprandStateScrambledSobol64 *devQStates,rk_state *devStates, retype *mcE,int reSampleTimes/*,int tidx*/){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N /*&& idx==tidx*/){
        // arrUcharMapper mask_adA(mask_ad,sz_tag);
        // mcE[idx]=drawDisIdx(s_n,gpp,devQStates+idx);
        // mcE[idx]=drawTau(25,devQStates+idx);
        // mcE[idx]=drawTau(25,devQStates);
        // float t=1;
        // draw_P_B_Tr(mcE+idx,35,1,&t,6 ,devQStates+idx);
        // mcE[idx]=drawE(3.0,6,devQStates+idx);        
        // mcE[idx]=drawA_fi_e(devStates+idx, 5, 0.7) ;
        // mcE[tidx]=drawA_fi_e(devStates, 5, 0.7) ;
        // mcE[idx]=drawJ_Si2Sj(gpp,s_n,2,devQStates+idx);
        // cuList<int> l1;
        // for (int ti=0;ti<10;ti++){
        //     l1.append(ti);
        // }
        // arrI64 a(10);
        // a<<0,0,7,7,0,2,5,6,7,3;
        // arrI hist(9);
        // binTimeHist(&hist, a,l1);
        // mcE[idx]=hist(9);
        // l1.freeList();      

        arrUcharMapper mask_adA(mask_ad+istart[idx],istop[idx]-istart[idx]);
        arrUcharMapper mask_ddA(mask_dd+istart[idx],istop[idx]-istart[idx]);
        arrI64Mapper times_msA(times_ms+istart[idx],istop[idx]-istart[idx]);        
        arrI64 burst_dd=mask_ddA.cast<int64_t>()*times_msA;
        arrI64 burst_ad=mask_adA.cast<int64_t>()*times_msA;
        for (int sampleTime=0;sampleTime<reSampleTimes;sampleTime++){
            int si=drawDisIdx(s_n,gpp,devQStates+idx);
            cuList<int> sidx;
            cuList<int64_t> bins;
            bins.append(start[idx]);
            sidx.append(si);            
            float mcSpendTime=0;
            matXfMapper matK(gpk,s_n,s_n);
            while (T[idx]>mcSpendTime){
                int sj=drawJ_Si2Sj(gpp,s_n,si,devQStates+idx);
                sidx.append(sj);
                mcSpendTime+=drawTau(matK(si,sj),devQStates+idx);
                si=sj;                
                if(mcSpendTime>=T[idx]){
                    bins.append(stop[idx]);
                }
                else{
                    bins.append(*(bins.at(0))+mcSpendTime/clk_p);
                }
            }            
            arrF f_ia(bins.len-1);
            binTimeHist(&f_ia,burst_ad,bins);
            arrF f_id(bins.len-1);
            binTimeHist(&f_id,burst_dd,bins);            
            arrI f_i(bins.len-1);
            arrF f_if(bins.len-1);
            f_if=(gamma-Dch2Ach)*f_id + (1-DexDirAem)*f_ia;
            arrF t_diff(bins.len-1);
            bins.diff(&t_diff);
            t_diff=t_diff*clk_p;
            for (int s_trans=0;s_trans<bins.len-1;s_trans++){

            }
            sidx.freeList();
            bins.freeList();
        }
    }
    
}

mc::mc(int id){
    devid=id;
    matK=NULL;matP=NULL;
    hpe=hpv=hpk=hpp=gpe=gpv=gpp=gpk=NULL;    
    devStates=NULL;
    devQStates=NULL;        
    CUDA_CHECK_RETURN(hipSetDevice(devid));
    hostVectors64=NULL;
    hostScrambleConstants64=NULL;
    devDirectionVectors64=NULL;
    devScrambleConstants64=NULL;
    
}
void mc::set_reSampleTimes(int t){
    reSampleTimes=t;
}

void mc::init_data_gpu(vector<int64_t>& start,vector<int64_t>& stop,
        vector<uint32_t>& istart,vector<uint32_t>& istop,
        vector<int64_t>& times_ms,
        vector<unsigned char>& mask_ad,vector<unsigned char>& mask_dd,
        vector<float>& T_burst_duration,vector<float>& SgDivSr,
        float& iclk_p,float& ibg_ad_rate,float& ibg_dd_rate){    
    clk_p=iclk_p;bg_ad_rate=ibg_ad_rate;bg_dd_rate=ibg_dd_rate;    
    sz_tag=mask_ad.size();                    
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hchi2, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_mask_ad, sizeof(unsigned char)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_mask_ad, mask_ad.data(), sizeof(unsigned char)*sz_tag, 
        hipMemcpyHostToDevice));    
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_mask_dd, sizeof(unsigned char)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_mask_dd, mask_dd.data(), sizeof(unsigned char)*sz_tag,hipMemcpyHostToDevice));
    sz_burst=start.size();         
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_start, sizeof(int64_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_start, start.data(), sizeof(int64_t)*sz_burst,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_stop, sizeof(int64_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_stop, stop.data(), sizeof(int64_t)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_istart, sizeof(uint32_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_istart, istart.data(), sizeof(uint32_t)*sz_burst,hipMemcpyHostToDevice));    
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_istop, sizeof(uint32_t)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_istop, istop.data(), sizeof(uint32_t)*sz_burst,hipMemcpyHostToDevice));            
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_times_ms, sizeof(int64_t)*sz_tag));
    CUDA_CHECK_RETURN(hipMemcpy(g_times_ms, times_ms.data(), sizeof(int64_t)*sz_tag,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_burst_duration, sizeof(float)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_burst_duration, T_burst_duration.data(), sizeof(float)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&g_SgDivSr, sizeof(float)*sz_burst));
    CUDA_CHECK_RETURN(hipMemcpy(g_SgDivSr, SgDivSr.data(), sizeof(float)*sz_burst,hipMemcpyHostToDevice));        
    CUDA_CHECK_RETURN(hipMalloc((void **)&gchi2, sizeof(float)));
}

void mc::run_kernel(int cstart,int cstop){  
    int N=cstop-cstart;
    int dimension=128;  
    dim3 threads = dim3(dimension, 1);
    int blocksCount = ceil(N / dimension);
    dim3 blocks  = dim3(blocksCount, 1);    
    CUDA_CHECK_RETURN(hipFree ( devStates));
    CUDA_CHECK_RETURN(hipFree ( devQStates));    
    CUDA_CHECK_RETURN(hipMalloc ( (void **)&devStates, N*sizeof (rk_state ) ));
    CUDA_CHECK_RETURN(hipMalloc ( (void **)&devQStates, N*sizeof( hiprandStateScrambledSobol64) ));    
    CUDA_CHECK_RETURN(hipFree (devDirectionVectors64));
    CUDA_CHECK_RETURN(hipFree (devScrambleConstants64));
    // CUDA_CHECK_RETURN
    (hiprandGetDirectionVectors64( &hostVectors64, 
        HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6));
    // CUDA_CHECK_RETURN
    (hiprandGetScrambleConstants64( &hostScrambleConstants64));              
    CUDA_CHECK_RETURN(hipMalloc((void **)&(devDirectionVectors64), 
    N * VECTOR_SIZE * sizeof(long long int)));        
    CUDA_CHECK_RETURN(hipMemcpy(devDirectionVectors64, hostVectors64,
    N * VECTOR_SIZE * sizeof(long long int), 
    hipMemcpyHostToDevice)); 
    CUDA_CHECK_RETURN(hipMalloc((void **)&(devScrambleConstants64), 
    N * sizeof(long long int)));
    CUDA_CHECK_RETURN(hipMemcpy(devScrambleConstants64, hostScrambleConstants64,
    N * sizeof(long long int), 
    hipMemcpyHostToDevice));
    setup_kernel <<<blocks, threads>>>(devStates, 0,/*time(NULL)*/ N ,
        devDirectionVectors64, devScrambleConstants64, devQStates);

    retype *mcE,*hmcE;
    CUDA_CHECK_RETURN(hipMalloc((void **)&mcE, N * sizeof(retype)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hmcE, N * sizeof(retype)));
    // int ti=0;
    // for( ;ti<N;ti++)
    mc_kernel<<<blocks, threads>>>(gchi2, g_start,g_stop,
        g_istart,g_istop,
        g_times_ms,
        g_mask_ad,g_mask_dd,
        g_burst_duration,g_SgDivSr,
        clk_p,bg_ad_rate,bg_dd_rate,sz_tag,sz_burst ,
        gpe,gpv,gpk,gpp,N,s_n,devQStates,devStates, mcE, reSampleTimes/*,ti*/);
    CUDA_CHECK_RETURN(hipMemcpy(hmcE, mcE,N * sizeof(retype), hipMemcpyDeviceToHost));        
    std::vector<retype> my_vector(hmcE, hmcE + N);
    for (int ip=0;ip<N;ip++)
        cout<<my_vector.at(ip)<<" ";
    cout<<endl;
    savehdf5("r.hdf5", "/r",my_vector);
    CUDA_CHECK_RETURN(hipFree(mcE));
    CUDA_CHECK_RETURN(hipHostFree(hmcE));
}

mc::~mc(){
    free_data_gpu();
    delete(matK);delete(matP);
}

void mc::free_data_gpu(){            
    CUDA_CHECK_RETURN(hipFree(g_mask_ad));
    CUDA_CHECK_RETURN(hipFree(g_mask_dd));
    CUDA_CHECK_RETURN(hipFree(g_start));
    CUDA_CHECK_RETURN(hipFree(g_stop));    
    CUDA_CHECK_RETURN(hipFree(g_istart));
    CUDA_CHECK_RETURN(hipFree(g_istop)); 
    CUDA_CHECK_RETURN(hipFree(g_times_ms));
    CUDA_CHECK_RETURN(hipFree(g_SgDivSr));
    CUDA_CHECK_RETURN(hipFree(g_burst_duration));

    // CUDA_CHECK_RETURN(hipFree(r_size));
    hipDeviceSynchronize();
    cout<<"rsize:"<<*hchi2<<endl;
    CUDA_CHECK_RETURN(hipHostFree(hchi2));
    CUDA_CHECK_RETURN(hipHostFree(hpe));
    CUDA_CHECK_RETURN(hipHostFree(hpv));
    CUDA_CHECK_RETURN(hipHostFree(hpp));
    CUDA_CHECK_RETURN(hipHostFree(hpk));
    CUDA_CHECK_RETURN(hipFree(gpe));
    CUDA_CHECK_RETURN(hipFree(gpv));
    CUDA_CHECK_RETURN(hipFree(gpp));
    CUDA_CHECK_RETURN(hipFree(gpk));        
}

bool mc::set_nstates(int n){
    s_n=n;
    bool r=true;
    CUDA_CHECK_RETURN(hipHostFree(hpe));
    CUDA_CHECK_RETURN(hipHostFree(hpv));
    CUDA_CHECK_RETURN(hipHostFree(hpp));
    CUDA_CHECK_RETURN(hipHostFree(hpk));
    CUDA_CHECK_RETURN(hipFree(gpe));
    CUDA_CHECK_RETURN(hipFree(gpv));
    CUDA_CHECK_RETURN(hipFree(gpp));
    CUDA_CHECK_RETURN(hipFree(gpk));    
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpe, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpv, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpp, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **)&hpk, n*n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpe, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpv, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpp, n*sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&gpk, n*n*sizeof(float)));    
    return r;
}

bool mc::set_params(vector<float>& args){
    int n=s_n;
    vecFloatMapper evargs(args.data(),n*n+n);    
    // cout<<evargs<<endl;
    eargs=evargs(seqN(0,n));
    float *peargs=eargs.data();
    kargs=evargs(seqN(n,n*n-n));    
    vargs=evargs(seqN(n*n,n));    
    float *pvargs=vargs.data();
    bool r=genMatK(&matK,n,kargs);
    //&matK不可修改，但是matK的值可以修改    
    r=r&&genMatP(&matP,matK);    
    // cout<<"p:"<<*matP<<endl;
    memcpy(hpe, peargs, sizeof(float)*n);
    memcpy(hpv, pvargs, sizeof(float)*n);
    memcpy(hpk, matK->data(), sizeof(float)*n*n);
    memcpy(hpp, matP->data(), sizeof(float)*n);
    CUDA_CHECK_RETURN(hipMemcpy(gpe,hpe,sizeof(float)*n,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpv,hpv, sizeof(float)*n,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpk,hpk, sizeof(float)*n*n,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpp,hpp, sizeof(float)*n,hipMemcpyHostToDevice));    
    return r;
}