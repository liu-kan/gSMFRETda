#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  hipStream_t stream1;  
  hipStreamCreate(&stream1);
  hipMemcpyAsync(d_x, x, N*sizeof(float), hipMemcpyHostToDevice,stream1);
  hipMemcpyAsync(d_y, y, N*sizeof(float), hipMemcpyHostToDevice,stream1);

  hipStream_t stream2;  
  hipStreamCreate(&stream2);
    
  // Perform SAXPY on 1M elements
//   saxpy<<<(N+255)/256, 256,0,stream1>>>(N, 2.0f, d_x, d_y);
hipStreamSynchronize(stream1);
  saxpy<<<(N+255)/256, 256,0,stream2>>>(N, 2.0f, d_x, d_y);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}