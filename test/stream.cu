#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include <rmm/detail/error.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>
#include <rmm/mr/device/default_memory_resource.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/managed_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include <rmm/mr/device/pool_memory_resource.hpp>
#include "rmm/detail/error.hpp"
#include "rmm/mr/device/cuda_memory_resource.hpp"
#include "rmm/mr/device/default_memory_resource.hpp"
#include "rmm/mr/device/device_memory_resource.hpp"
using Pool = rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource>;

#define N (1024*1024) 
#define FULL_DATA_SIZE (N*20)
__global__ void kernel( int *a, int *b, int *c ) 
{ 
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
         int idx1 = (idx + 1) % 256; 
         int idx2 = (idx + 2) % 256; 
         float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f; 
         float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f; 
        //  printf("a=%d\n",a[idx]);
         c[idx] = (as + bs) / 2;
    }
}
int main( void ) { 
    hipDeviceProp_t prop; 
    int whichDevice;
    checkCudaErrors( hipGetDevice( &whichDevice ) );
    checkCudaErrors( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) { 
        printf( "Device will not handle overlaps, so no " 
        "speed up from streams\n" );
        return 0; 
    }
        hipEvent_t start, stop;
        float elapsedTime;
        // start the timers
        checkCudaErrors( hipEventCreate( &start ) ); checkCudaErrors( hipEventCreate( &stop ) );
        checkCudaErrors( hipEventRecord( start, 0 ) );
        // initialize the streams 
        hipStream_t stream0, stream1;
        checkCudaErrors( hipStreamCreate( &stream0 ) );
        checkCudaErrors( hipStreamCreate( &stream1 ) );
        int *host_a, *host_b, *host_c; 
        int *dev_a0, *dev_b0, *dev_c0; 
        int *dev_a1, *dev_b1, *dev_c1; 
        auto const max_pool =
        static_cast<std::size_t>(rmm::mr::detail::available_device_memory());
        Pool *mr=new Pool{rmm::mr::get_current_device_resource(),static_cast<std::size_t>(max_pool*0.75),
          static_cast<std::size_t>(max_pool*0.9)};

        //GPU buffers for stream1
        // allocate the memory on the GPU 
        // checkCudaErrors( hipMalloc( (void**)&dev_a0, N * sizeof(int) ) );
        // rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource();
        dev_a0= mr->allocate(N * sizeof(int), stream0);

        // rmm::device_buffer buff(N * sizeof(int), stream0);
        checkCudaErrors(hipStreamSynchronize(stream0));
        // dev_a0=(int *)buff.data();
        checkCudaErrors( hipMalloc( (void**)&dev_b0, N * sizeof(int) ) );
        checkCudaErrors( hipMalloc( (void**)&dev_c0, N * sizeof(int) ) );
checkCudaErrors( hipMalloc( (void**)&dev_a1, N * sizeof(int) ) );
checkCudaErrors( hipMalloc( (void**)&dev_b1, N * sizeof(int) ) );
checkCudaErrors( hipMalloc( (void**)&dev_c1, N * sizeof(int) ) );
// allocate page-locked memory, used to stream 
checkCudaErrors( hipHostAlloc( (void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault ) );
checkCudaErrors( hipHostAlloc( (void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault ) );
checkCudaErrors( hipHostAlloc( (void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault ) );
for (int i=0; i<FULL_DATA_SIZE; i++) {
     host_a[i] = i; host_b[i] = rand();
}
// now loop over full data, in bite-sized chunks 
for (int i=0; i<FULL_DATA_SIZE; i+= N*2) { // copy the locked memory to the device, async 
    checkCudaErrors( hipMemcpyAsync( dev_a0, host_a+i, N * sizeof(int),
    hipMemcpyHostToDevice, stream0 ) );
    checkCudaErrors( hipMemcpyAsync( dev_b0, host_b+i, N * sizeof(int),
    hipMemcpyHostToDevice, stream0 ) );
    kernel<<<N/256,256,0,stream0>>>( dev_a0, dev_b0, dev_c0 );
    // copy the data from device to locked memory 
    checkCudaErrors( hipMemcpyAsync( host_c+i, dev_c0, N * sizeof(int),
    hipMemcpyDeviceToHost,
    stream0 ) );

    // copy the locked memory to the device, async 
    checkCudaErrors( hipMemcpyAsync( dev_a1, host_a+i+N, N * sizeof(int),
hipMemcpyHostToDevice, stream1 ) );
checkCudaErrors( hipMemcpyAsync( dev_b1, host_b+i+N, N * sizeof(int),
hipMemcpyHostToDevice,
stream1 ) );
kernel<<<N/256,256,0,stream1>>>( dev_a1, dev_b1, dev_c1 ); // copy the data from device to locked memory 
checkCudaErrors( hipMemcpyAsync( host_c+i+N, dev_c1, N * sizeof(int),
hipMemcpyDeviceToHost,
stream1 ) );
}
checkCudaErrors( hipStreamSynchronize( stream0 ) ); checkCudaErrors( hipStreamSynchronize( stream1 ) );

checkCudaErrors( hipEventRecord( stop, 0 ) ); checkCudaErrors( hipEventSynchronize( stop ) ); 
checkCudaErrors( hipEventElapsedTime( &elapsedTime, start, stop ) );
printf( "Time taken: %3.1f ms\n", elapsedTime );
// cleanup the streams and memory 
checkCudaErrors( hipHostFree( host_a ) ); checkCudaErrors( hipHostFree( host_b ) );
checkCudaErrors( hipHostFree( host_c ) );

// checkCudaErrors( hipFree( dev_a0 ) ); 
// rmm::device_memory_resource::deallocate(dev_a0, N * sizeof(int), stream0);
mr->deallocate(dev_a0, N * sizeof(int), stream0);
delete(mr);
checkCudaErrors( hipFree( dev_b0 ) ); checkCudaErrors( hipFree( dev_c0 ) ); 
checkCudaErrors( hipFree( dev_a1 ) ); checkCudaErrors( hipFree( dev_b1 ) ); checkCudaErrors( hipFree( dev_c1 ) );
 checkCudaErrors( hipStreamDestroy( stream0 ) ); checkCudaErrors( hipStreamDestroy( stream1 ) );
return 0;
}