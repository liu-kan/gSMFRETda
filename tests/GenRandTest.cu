#include <hip/hip_runtime_api.h>
#include "GenRandTest.hpp"
#include "binom.cuh"
#include "gen_rand.cuh"

void GenRandTest::SetUp(){
  printf("GenRandTest SetUp()\n");
  int nDevices=0;
  hipGetDeviceCount(&nDevices);
  ASSERT_GE(nDevices, 1) << "You need at least 1 NVIDIA GPU to run tests!";
  CUDA_CHECK_RETURN(hipSetDevice(nDevices-1));
  // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, mc_kernel, 0, 0);
  // blockSize=128;
  // printf("blockSize = %d\n", blockSize);
}

void GenRandTest::TearDown() {
  hipDeviceReset();
  std::cout<<"hipDeviceReset done!\n";
}
void GenRand::init_randstate(int N){
  CURAND_CALL(hiprandGetDirectionVectors64(
              &hostVectors64, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6));
  CURAND_CALL(hiprandGetScrambleConstants64(&hostScrambleConstants64));  
  int NN;
  NN = N * reSampleTimes;
  gridSize = (NN + blockSize - 1) / blockSize;
  CUDA_CHECK_RETURN(hipMalloc((void **)&devStates, NN * sizeof(rk_state)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&devQStates, NN * sizeof(hiprandStateScrambledSobol64)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&devDirectionVectors64, 
                                NN * VECTOR_SIZE * sizeof(long long int)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&devScrambleConstants64, 
                                NN * sizeof(long long int)));
  int n = 0;
  int tNN = NN;
  while (tNN > 0) {
      int size = (tNN > 20000) ? 20000 : tNN;
      unsigned long long int *buf = devScrambleConstants64;
      CUDA_CHECK_RETURN(hipMemcpy(buf + n * 20000, hostScrambleConstants64,
                                        size * sizeof(unsigned long long int),
                                        hipMemcpyHostToDevice));
      // std::cout << "n = " << n << ", size = " << size << std::endl;
      buf = devDirectionVectors64;
      CUDA_CHECK_RETURN(
          hipMemcpy(buf + n * 20000 * sizeof(hiprandDirectionVectors64_t) /
                                    sizeof(unsigned long long int),
                          hostVectors64, size * sizeof(hiprandDirectionVectors64_t),
                          hipMemcpyHostToDevice));
      tNN -= size;
      n++;
  }
  setup_kernel<<<gridSize, blockSize>>>(
      devStates, 0, /*time(NULL)*/ NN, devDirectionVectors64,
      devScrambleConstants64, devQStates);
  std::cout<<"setup_kernel done!\n";
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
}
