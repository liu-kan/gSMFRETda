#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "GenRandTest.hpp"
#include "binom.cuh"
#include "gen_rand.cuh"
#include <random>
#include "hist.hpp"
#include <vector>
#include <dlib/statistics.h>
void GenRandTest::SetUp(){
  printf("GenRandTest SetUp()\n");
  int nDevices=0;
  hipGetDeviceCount(&nDevices);
  ASSERT_GE(nDevices, 1) << "You need at least 1 NVIDIA GPU to run tests!";  
}

void GenRandTest::TearDown() {
    printf("GenRandTest TearDown()\n");
}

GenRand::GenRand() {
    randstateN = 0;
    CUDA_CHECK_RETURN(hipSetDevice(0));
    devDirectionVectors64=NULL;
    devScrambleConstants64=NULL;   
    int_res=NULL;
}
GenRand::~GenRand() {
    hipDeviceReset();
    std::cout << "hipDeviceReset done!\n";
}
/**
 * @brief init randstate
 * 
 * @param N Number of samples
 */
void GenRand::init_randstate(int N){
  // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, setup_kernel, 0, 0);
  blockSize=256;
  std::cout<<"blockSize: "<<blockSize<<std::endl;
  CURAND_CALL(hiprandGetDirectionVectors64(
              &hostVectors64, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6));
  CURAND_CALL(hiprandGetScrambleConstants64(&hostScrambleConstants64));  
  int NN;
  NN = N;  
  gridSize = (NN + blockSize - 1) / blockSize;
  CUDA_CHECK_RETURN(hipMalloc((void **)&devStates, NN * sizeof(rk_state)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&devQStates, NN * sizeof(hiprandStateScrambledSobol64)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&devDirectionVectors64, 
                                NN * VECTOR_SIZE * sizeof(long long int)));
  CUDA_CHECK_RETURN(hipMalloc((void **)&devScrambleConstants64, 
                                NN * sizeof(unsigned long long int)));
  int n = 0;
  int tNN = NN;
  while (tNN > 0) {
      int size = (tNN > 20000) ? 20000 : tNN;
      std::cout << "n= " << n << std::endl;
      CUDA_CHECK_RETURN(hipMemcpy(devScrambleConstants64 + n * 20000, hostScrambleConstants64,
                                        size * sizeof(unsigned long long),
                                        hipMemcpyHostToDevice));
      // std::cout << "n = " << n << ", size = " << size << std::endl;
      //buf = devDirectionVectors64;
      CUDA_CHECK_RETURN(
          hipMemcpy(devDirectionVectors64 + n * 20000 * sizeof(hiprandDirectionVectors64_t) /
                                    sizeof(unsigned long long),
                          hostVectors64, size * sizeof(hiprandDirectionVectors64_t),
                          hipMemcpyHostToDevice));
      tNN -= size;
      n++;
  }
  setup_kernel<<<gridSize, blockSize>>>(
      devStates, 0, /*time(NULL)*/ NN, devDirectionVectors64,
      devScrambleConstants64, devQStates);
  std::cout<<"setup_kernel done!\n";
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
}
void GenRand::init_mem(int N,int n){
    hipError_t e = hipMalloc((void**)&gp, n * sizeof(float));
    ASSERT_EQ(e, hipSuccess) << "hipMalloc failed!";
    e = hipMalloc((void **)&int_res, N * sizeof(int));
    ASSERT_EQ(e, hipSuccess) << "hipMalloc failed!";
    randstateN = N;
}
void GenRand::free_mem() {    
    std::cout << " devDirectionVectors64: " << static_cast<void*>(devDirectionVectors64) << "\n";
    CUDA_CHECK_RETURN(hipFree(devDirectionVectors64));
    CUDA_CHECK_RETURN(hipFree(devScrambleConstants64));
    if (randstateN > 0) {
        CUDA_CHECK_RETURN(hipFree(int_res));        
    }

}

__global__ void
test_drawDisIdx_kernel(int n,float* p, int N, int* int_res,
          hiprandStateScrambledSobol64* devQStates) 
{
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < N) {
      int_res[tidx] = drawDisIdx(n, p, devQStates + tidx);
  }
}

/**
 * @brief 
 * 
 * @param n Number of states use in hist
 */
void GenRand::test_drawDisIdx(int n){
  // using namespace boost::histogram;
  std::random_device rd;  //Will be used to obtain a seed for the random number engine
  std::mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
  std::uniform_real_distribution<float> dis(0.0, 1.0);

  int scount = n * 30;
  float *rawp=new float[scount];
  int* res_c = new int[randstateN];
  float* p = new float[n];
  for (int i = 0; i < scount; i++) {
    rawp[i]=dis(gen);
  }  
  std::ostringstream os;
  getoss(rawp, scount, n,os,p);
  std::vector<float> tp(p, p+n);
  CUDA_CHECK_RETURN(hipMemcpy(gp, p,n * sizeof(float),hipMemcpyHostToDevice));
  std::cout << os.str() << std::flush;  
  test_drawDisIdx_kernel<<<gridSize, blockSize>>>(n,gp,randstateN,int_res,devQStates);
  CUDA_CHECK_RETURN(hipMemcpy(res_c, int_res, randstateN * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CHECK_RETURN(hipDeviceSynchronize());  
  os.str("");
  os.clear();
  getoss_i(res_c, randstateN, n, os, p);
  std::cout << os.str() << std::flush;
  std::vector<float> sp(p, p + n);  
  float r2 = dlib::r_squared(sp, tp);
  std::cout << "r2: " << r2 << std::endl;
  if(randstateN>10000)
    EXPECT_GE(r2, 0.78);
  delete[] p;
  delete[] rawp;
  delete[] res_c;
  
}